
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <unistd.h>
#include <math.h>
#include <time.h>

// Maximum value of the matrix element
#define MAX 100

#define MAX_ITER 100
#define ERROR 0.000001

// Generate a random float number with the maximum value of max
float rand_float(int max)
{
    return ((float)rand() / (float)(RAND_MAX)) * max;
}

// Calulates the space for the matrices (bytes)
int calc_mem_size(int n, int m)
{
    return (n * m * sizeof(float));
}

// Calculates the sum of a given flat matrix (array)
float calc_mat_sum(float **mat, int n, int m)
{

    float sum = 0.0f;
    for (long i = 0; i < (n * m); i++)
    {
        sum += (*mat)[i];
    }

    return sum;
}

// Allocate 2D matrix in the host
void alloc_host_matrix(float **mat, int n, int m, bool must_init)
{

    *mat = (float *)malloc(n * m * sizeof(float));

    // In case of initializing the matrix with the initial values
    if (must_init)
    {
        for (int i = 0; i < (n * m); i++)
        {
            (*mat)[i] = rand_float(MAX);
        }
    }
}

// Allocate 2D matrix in the device
void alloc_dev_matrix(float **mat, int n, int m)
{
    size_t memSize = (n * m * sizeof(float));
    hipMalloc(&mat, memSize);
}

// Write the time results into a CSV file
void write_to_file(int n, int num_blocks, int num_threads, float total_time, float exec_time)
{

    FILE *f;
    char *file_name = "./out/results.csv";

    if (access(file_name, F_OK) == -1)
    {
        f = fopen(file_name, "a");
        fprintf(f, "Matrix size;Blocks;Threads per block;Total time;Operations time;\n");
    }
    else
    {
        f = fopen(file_name, "a");
    }

    fprintf(f, "%d;%d;%d;%f;%f;\n", n, num_blocks, num_threads, total_time, exec_time);
    fclose(f);
}

// Solver (executed by each thread)
__global__ void solver(float **mat, float **mat_diff, int n)
{

    // Posição original que esta thread está atribuída
    int i_org = (blockDim.x * blockIdx.x) + threadIdx.x;

    // Posição real que esta thread irá calcular
    int i = i_org;
    i = i + n; // VIP: Os threads devem evitar a primeira linha
    i = i + 1; // VIP: As threads devem evitar a primeira coluna

    // Caso a tenha tenha sobrado
    if (i >= ((n * n) - n - 1))
    {
        return;
    }

    float temp;
    float diff = 0;
    int cnt_iter = 0;

    const int pos_up = i - n;
    const int pos_do = i + n;
    const int pos_le = i - 1;
    const int pos_ri = i + 1;

    while (cnt_iter < MAX_ITER)
    {

        temp = (*mat)[i];
        (*mat)[i] = 0.2 * ((*mat)[i] + (*mat)[pos_le] + (*mat)[pos_up] + (*mat)[pos_ri] + (*mat)[pos_do]);

        // A ÚLTIMA diferença entre o valor anterior e o novo valor é armazenada
        diff = abs((*mat)[i] - temp);
        cnt_iter++;
    }

    // Finalmente, a diferença é armazenada em sua célula correspondente
    // VIP: Use '=' e não '+=' para evitar valores diferentes de zero na primeira chamada de função
    (*mat_diff)[i_org] = diff;
}

int main(int argc, char *argv[])
{

    if (argc < 3)
    {
        printf("Passe dois parâmetros:\n");
        printf("\t matrix_size: Adicione 2 a uma potência de 2\n");
        printf("\t Melhor uma potência de 2 (e.g. : 16, 32, 64)\n");
        exit(1);
    }

    int n = atoi(argv[1]);
    int threads_per_block = atoi(argv[2]);
    printf("Tamanho da matriz = %d\n", n);
    printf("Threads por bloco = %d\n", threads_per_block);

    // Start recording the time
    clock_t i_total_t = clock();

    float *host_mat_vals;
    float *host_mat_diff;
    float *dev_mat_vals;
    float *dev_mat_diff;

    // Allocating matrices space both in host and device
    alloc_host_matrix(&host_mat_vals, n, n, true);
    alloc_host_matrix(&host_mat_diff, n - 2, n - 2, false);
    alloc_dev_matrix(&dev_mat_vals, n, n);
    alloc_dev_matrix(&dev_mat_diff, n - 2, n - 2);

    // Calculating the memory size for allocating (bytes)
    size_t all_mat_size = calc_mem_size(n, n);
    size_t core_mat_size = calc_mem_size(n - 2, n - 2);

    // Passing data from host to device
    hipMemcpy(dev_mat_vals, host_mat_vals, all_mat_size, hipMemcpyHostToDevice);

    // Calculate the number of threads to launch (1 per core cell)
    int core_dim = (n - 2) * (n - 2);

    // Given a constant number of threads per block, determine the blocks
    int num_blocks = (int)ceil(core_dim / threads_per_block);
    dim3 dimGrid(num_blocks);
    dim3 dimBlock(threads_per_block);

    // Time before the execution
    clock_t i_exec_t = clock();

    while (true)
    {

        // Make all the threads synchronous
        solver<<<dimGrid, dimBlock>>>(&dev_mat_vals, &dev_mat_diff, n);

        // The ThreadSynchronize would be neccesary in case Memcpy is not done
        // However, as it is called later on, the following line is commented
        // cudaThreadSynchronize();

        // Passing the differential data back from the device to the host
        hipMemcpy(host_mat_diff, dev_mat_diff, core_mat_size, hipMemcpyDeviceToHost);

        // Breaks in case of reaching the ERROR threshold
        float diffs_sum = calc_mat_sum(&host_mat_diff, n - 2, n - 2);
        float diffs_mean = diffs_sum / (n - 2) / (n - 2);
        if (diffs_mean < ERROR)
        {
            break;
        }
    }

    // Time before the execution
    clock_t f_exec_t = clock();

    // Passing data back from the device to the host
    hipMemcpy(host_mat_vals, dev_mat_vals, all_mat_size, hipMemcpyDeviceToHost);

    // Finally, the matrices are freed
    hipFree(dev_mat_vals);
    hipFree(dev_mat_diff);
    free(host_mat_vals);
    free(host_mat_diff);

    // Finish recording the time
    clock_t f_total_t = clock();

    float total_time = (float)(f_total_t - i_total_t) / CLOCKS_PER_SEC;
    float exec_time = (float)(f_exec_t - i_exec_t) / CLOCKS_PER_SEC;
    printf("Tempo total: %f\n", total_time);
    printf("Tempo de operação: %f\n", exec_time);

    write_to_file(n, num_blocks, threads_per_block, total_time, exec_time);
    return 0;
}